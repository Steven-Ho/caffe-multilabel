#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layer.hpp"
#include "caffe/util/math_functions.hpp"
#include "caffe/vision_layers.hpp"

namespace caffe {

template <typename Dtype>
__global__ void HierarchicalSoftmaxLossForwardGPU(const int nthreads,
          const Dtype* prob_data, const Dtype* label, Dtype* loss,
          const int num, const int dim, const int spatial_dim,
          const bool has_ignore_label_, const int ignore_label_,
          Dtype* counts, int label_num_, const Dtype* split_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    // const int s = index % spatial_dim;
    const int channels = dim / spatial_dim;
    counts[index] = 0;
    for (int i = 0; i < channels; i++) {
      loss[index] = 0;
      const int label_value = static_cast<int>(label[n * channels + i]);
      if (label_value == 1) {
        loss[index] -= log(max(prob_data[n * channels + i],
                      Dtype(FLT_MIN)));
        counts[index] += 1;
      }
    }
  }
}

template <typename Dtype>
void HierarchicalSoftmaxWithLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  softmax_layer_->Forward(softmax_bottom_vec_, softmax_top_vec_);
  const Dtype* prob_data = prob_.gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const int dim = prob_.count() / outer_num_;
  const int nthreads = outer_num_ * inner_num_;
  const Dtype* split_data = split_.gpu_data();
  // Since this memory is not used for anything until it is overwritten
  // on the backward pass, we use it here to avoid having to allocate new GPU
  // memory to accumulate intermediate results in the kernel.
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();
  // Similarly, this memory is never used elsewhere, and thus we can use it
  // to avoid having to allocate additional GPU memory.
  Dtype* counts = prob_.mutable_gpu_diff();
  // NOLINT_NEXT_LINE(whitespace/operators)
  HierarchicalSoftmaxLossForwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
      CAFFE_CUDA_NUM_THREADS>>>(nthreads, prob_data, label, loss_data,
      outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts, label_num_, split_data);
  Dtype loss;
  caffe_gpu_asum(nthreads, loss_data, &loss);
  if (normalize_) {
    Dtype count;
    caffe_gpu_asum(nthreads, counts, &count);
    loss /= count;
  } else {
    loss /= outer_num_;
  }
  top[0]->mutable_cpu_data()[0] = loss;
  if (top.size() == 2) {
    top[1]->ShareData(prob_);
  }
}

template <typename Dtype>
__global__ void HierarchicalSoftmaxLossBackwardGPU(const int nthreads, const Dtype* top,
          const Dtype* label, Dtype* bottom_diff, const int num, const int dim,
          const int spatial_dim, const bool has_ignore_label_,
          const int ignore_label_, Dtype* counts, int label_num_, const Dtype* split_data) {
  const int channels = dim / spatial_dim;

  CUDA_KERNEL_LOOP(index, nthreads) {
    const int n = index / spatial_dim;
    // const int s = index % spatial_dim;

    counts[index] = 0;
    for (int i = 0; i < label_num_; i++) {
      int start = split_data[2 * i];
      int end = split_data[2 * i + 1];
      bool flag = false;
      int idx = 0;
      for (int k = start; k <= end; k++) {
        if (label[n * channels + k] == 1) {
          flag = true;
          idx = k;
          break;
        }
      }
      if (flag) {
        bottom_diff[n * dim + idx] -= 1;
        counts[index] += 1;
      } else {
        for (int k = start; k <= end; k++) {
          bottom_diff[n * channels + k] = 0;
        }
      }
    }
  }
}

template <typename Dtype>
void HierarchicalSoftmaxWithLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type()
               << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* top_data = top[0]->gpu_data();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    const Dtype* label = bottom[1]->gpu_data();
    const int dim = prob_.count() / outer_num_;
    const int nthreads = outer_num_ * inner_num_;
    const Dtype* split_data = split_.gpu_data();
    // Since this memory is never used for anything else,
    // we use to to avoid allocating new GPU memory.
    Dtype* counts = prob_.mutable_gpu_diff();
    // NOLINT_NEXT_LINE(whitespace/operators)
    HierarchicalSoftmaxLossBackwardGPU<Dtype><<<CAFFE_GET_BLOCKS(nthreads),
        CAFFE_CUDA_NUM_THREADS>>>(nthreads, top_data, label, bottom_diff,
        outer_num_, dim, inner_num_, has_ignore_label_, ignore_label_, counts, label_num_, split_data);
    Dtype count;
    caffe_gpu_asum(nthreads, counts, &count);

    const Dtype loss_weight = top[0]->cpu_diff()[0];
    if (normalize_) {
      caffe_gpu_scal(prob_.count(), loss_weight / count, bottom_diff);
    } else {
      caffe_gpu_scal(prob_.count(), loss_weight / outer_num_, bottom_diff);
    }
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(HierarchicalSoftmaxWithLossLayer);

}  // namespace caffe
